#include <iostream>
#include <hip/hip_runtime.h>

__global__ void vectorAdd(const float* A, const float* B, float* C, int N) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < N) {
        C[i] = A[i] + B[i];
    }
}

int main() {
    const int N = 10;
    float A[N], B[N], C[N];

    float *d_a, *d_b,*d_c;
    //allocate memory for device
    hipMalloc(&d_a,N*sizeof(float));
    hipMalloc(&d_b,N*sizeof(float));
    hipMalloc(&d_c,N*sizeof(float));
    //initialize host array
    hipMemcpy(d_a,A,N*sizeof(float),hipMemcpyHostToDevice);
    hipMemcpy(d_b,B,N*sizeof(float),hipMemcpyHostToDevice);

    int blocksize=256;
    int gridsize=ceil(N/blocksize);

    vectorAdd<<<gridsize,blocksize>>>(d_a,d_b,d_c,N);
    hipMemcpy(C,d_c,N*sizeof(float),hipMemcpyDeviceToHost);
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
}